#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <hip/hip_runtime.h>

#define IN_FILE_NAME "lena.png"
#define GRAY_FILE_NAME "CUDA_GRAY.bmp"
#define XFILTERED_FILE_NAME "CUDA_XFILTERED.bmp"
#define YFILTERED_FILE_NAME "CUDA_YFILTERED.bmp"
#define RES_FILE_NAME "CUDA_ZRESULT.bmp"

#define MASK_DIM 3
#define MASK_LENGTH MASK_DIM*MASK_DIM
#define MASK_RADIUS MASK_DIM/2
const char sobelx[9] = { -1, 0, 1, -2, 0, 2, -1, 0, 1 };
const char sobely[9] = { 1, 2, 1, 0, 0, 0, -1, -2, -1 };


//returns brightness of a pixel. If number is out of bounds, returns value of border pixel.
int getBrightness(int x, int y, unsigned char *img, int imgWidth, int imgHeight){
	if (x < 0) x = 0;
	else if (x >= imgWidth) x = imgWidth - 1;
	if (y < 0) y = 0;
	else if (y >= imgHeight) y = imgHeight - 1;
	return img[y*imgWidth + x];
}

__global__ void
copyArrays(const unsigned char *imgDev, unsigned char *resultDev, int width, int height){
	
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < width*height)
	{
		resultDev[i] = imgDev[i];
	}
	__syncthreads();
}

__global__ void

copyArrays2(const unsigned char *imgDev, unsigned char *resultDev, int width, int height){
	
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int x = index%width;
	int y = index/width;
	for (int y = 0; y < width; y++){
			for (int x = 0; x < height; x++){
				resultDev[y*width+x] = 255-imgDev[y*width+x];
			}
		}
	__syncthreads();
}

__global__ void
filter(const unsigned char *imgDev, unsigned char *resultDev, int width, int height, const char* mask, int sizeOfMask){

	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int x = index%width;
	int y = index/width;

	int res = 0;
	int maskIndex = 0;
	int dx = 0;
	int dy = 0;
	if(index < width*height){
		for (int yi = y - MASK_RADIUS; yi <= y + MASK_RADIUS; yi++){
			for (int xi = x - MASK_RADIUS; xi <= x + MASK_RADIUS; xi++){
				dx = xi;
				dy = yi;
				dx = max(0, dx);
				dx = min(dx, width - 1);
				dy = max(0, dy);
				dy = min(dy, height - 1);
				res += mask[maskIndex] * imgDev[dy*width + dx];
				maskIndex++;
			}
		}
		if (res < 0) res = -res;
		resultDev[index] = (unsigned char)res;
	}
	__syncthreads();
}

void filterSeq(int x, int y, const unsigned char *img, unsigned char **result, int width, int height, const char* mask, int sizeOfMask){

	//load image?

	int i = y*width + x;

	int res = 0;
	int maskIndex = 0;
	int dx = 0;
	int dy = 0;
	for (int yi = y - MASK_RADIUS; yi <= y + MASK_RADIUS; yi++){
		for (int xi = x - MASK_RADIUS; xi <= x + MASK_RADIUS; xi++){
			dx = xi;
			dy = yi;
			dx = max(0, dx);
			dx = min(dx, width - 1);
			dy = max(0, dy);
			dy = min(dy, height - 1);
			res += mask[maskIndex] * img[dy*width + dx];
			maskIndex++;
		}
	}
	if (res < 0) res = -res;
	(*result)[i] = (unsigned char)res;
	
	return;
}

void countGradient(int x, int y, const unsigned char *xfiltered, const unsigned char *yfiltered, unsigned char **result, int width, int height){
	
	int i = y*width + x;
	int res = 0;
	
	if (xfiltered[i] > yfiltered[i]){
		res = xfiltered[i] - yfiltered[i];
	}else{
		res = yfiltered[i] - xfiltered[i];
	}
	
	(*result)[i] = res;
	//(*result)[i] = res > threshold ? 255 : 0;
	return;
}

int main(int argc, char *argv[]) {

	hipError_t err = hipSuccess;

	printf("MASK_RADIUS = %d\n", MASK_RADIUS);

	int width, height, bpp;
	printf("Loading image file\n");
	unsigned char* img = stbi_load(IN_FILE_NAME, &width, &height, &bpp, STBI_grey);
	if (img == 0){
		printf("Failed to load image\n");
		system("PAUSE");
		return 0;
	}
	else{
		printf("Load complete!\n");
	}

	int size = width*height;
	int threadsPerBlock = 256;
	int blocksPerGrid = size/threadsPerBlock;
	printf("TOTAL PIXELS  %d!\n", width*height);
	printf("TOTAL THREADS %d!\n", blocksPerGrid*threadsPerBlock);

	size_t allocSize = size*sizeof(unsigned char);
	printf("I want to alloc memes!\n");
	unsigned char* xfiltered = (unsigned char*)malloc(allocSize);
	printf("X filtered mem aloced!\n");
	unsigned char* yfiltered = (unsigned char*)malloc(allocSize);
	printf("Y filtered mem aloced!\n");
	unsigned char* result = (unsigned char*)malloc(allocSize);
	printf("Result mem aloced!\n");
	//----------------------------------------------------------------------CUDA MALLOC
	unsigned char *imgDev = NULL;
	unsigned char *xfilteredDev = NULL;
	unsigned char *yfilteredDev = NULL;
	unsigned char *resultDev = NULL;
	err = hipMalloc((void **)&imgDev, allocSize);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device image memory (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	printf("device image mem aloced!\n");
	err = hipMalloc((void **)&xfilteredDev, allocSize);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device xfiltered memory (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	printf("device xfiltered mem aloced!\n");
	err = hipMalloc((void **)&yfilteredDev, allocSize);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device yfiltered memory (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	printf("device yfiltered mem aloced!\n");
	err = hipMalloc((void **)&resultDev, allocSize);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device image memory (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	printf("device result mem aloced!\n");
	//----------------------------------------------------------------------
	printf("Copying input data from the host memory to the CUDA device\n");
	err = hipMemcpy(imgDev, img, allocSize, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy image from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}else{
		printf("Image copied to device!\n");
	}

	//***********************************************************************************************************************************************************
	/*filter<<<blocksPerGrid, threadsPerBlock>>>(imgDev, xfilteredDev, width, height, sobelx, MASK_LENGTH);
	err = hipGetLastError();
	if (err != hipSuccess){
		fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	filter<<<blocksPerGrid, threadsPerBlock>>>(imgDev, yfilteredDev, width, height, sobely, MASK_LENGTH);
	err = hipGetLastError();
	if (err != hipSuccess){
		fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}*/
	copyArrays2<<<blocksPerGrid, threadsPerBlock>>>(imgDev, resultDev, width, height);
	err = hipGetLastError();
	if (err != hipSuccess){
		fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	//SEQUENTIAL CPU CODE HERE
	/*for (int y = 0; y < height ; y++){
		for (int x = 0; x < width; x++){
			//if (x == 0) printf("|\n");
			//printf("%d ", getBrightness(x, y, img, width, height));
			filterSeq(x, y, img, &xfiltered, width, height, sobelx, MASK_LENGTH);	
			filterSeq(x, y, img, &yfiltered, width, height, sobely, MASK_LENGTH);
			countGradient(x, y, xfiltered, yfiltered, &result, width, height);
		}
		if (y % 100 == 0) printf("\rFiltering %.2f %%", 100.0*y / height);
	}
	printf("\rFiltering 100.00%%");
	*/

	//----------------------------------------------------------------------------------------CUDA MEM COPY DEVICE TO HOST
	err = hipMemcpy(result, resultDev, allocSize, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy result from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}else{
		printf("result copied to device!\n");
	}
	err = hipMemcpy(xfiltered, xfilteredDev, allocSize, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy xfiltered from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}else{
		printf("xfiltered copied to device!\n");
	}
	err = hipMemcpy(yfiltered, yfilteredDev, allocSize, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy yfiltered from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}else{
		printf("yfiltered copied to device!\n");
	}
	

	//saving images to HDD
	printf("\nCreating gray file\n");
	stbi_write_bmp(GRAY_FILE_NAME, width, height, STBI_grey, img);
	printf("\nCreating X filtered file");
	stbi_write_bmp(XFILTERED_FILE_NAME, width, height, STBI_grey, xfiltered);
	printf("\nCreating Y filtered file");
	stbi_write_bmp(YFILTERED_FILE_NAME, width, height, STBI_grey, yfiltered);
	printf("\nCreating result file");
	stbi_write_bmp(RES_FILE_NAME, width, height, STBI_grey, result);
	
	err = hipFree(xfilteredDev);
	err = hipFree(yfilteredDev);
	err = hipFree(resultDev);
	err = hipDeviceReset();
	printf("\nEnd of program!\n");
	stbi_image_free(img);	
	system("PAUSE");
	return 0;
}
